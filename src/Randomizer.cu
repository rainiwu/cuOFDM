#include "hip/hip_runtime.h"
#include "Randomizer.cuh"

__global__ void cuInitRand(hiprandState *aState) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(clock() + tid, tid, 0, &aState[tid]); 
}

__global__ void cuApplyRand(hipComplex* inBuff, hipComplex* outBuff, hiprandState *aState) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // generate rand
  float randx = hiprand_normal(&aState[tid]);
  float randy = hiprand_normal(&aState[tid]);

  randx = randx * ((float)BATCH_SIZE / 2048) * 2/3;
  randy = randy * ((float)BATCH_SIZE / 2048) * 2/3;

  // apply to signal
  outBuff[tid].x = inBuff[tid].x + randx;
  outBuff[tid].y = inBuff[tid].y + randy;
}

void initRand(hiprandState *aState) {
  cuInitRand<<<BATCH_SIZE/RAND_TNUM, RAND_TNUM>>>(aState);
}

void applyRand(hipComplex* inBuff, hipComplex* outBuff, hiprandState *aState, hipStream_t *aStream) {
  cuApplyRand<<<BATCH_SIZE/RAND_TNUM, RAND_TNUM, 0, *aStream>>>(inBuff, outBuff, aState);
}
