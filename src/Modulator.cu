#include "hip/hip_runtime.h"
#include "Modulator.cuh"
#include <hipfft/hipfft.h>

__global__ void modulate(uint8_t* dBits, hipComplex* dMods, hipComplex* dMap) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  dMods[tid] = dMap[dBits[tid]];

}

void process(uint8_t* inBuff, hipComplex* outBuff, hipComplex* myMap, hipfftHandle* myPlan, hipStream_t* aStream) {
  modulate<<<BATCH_SIZE/MOD_TNUM, MOD_TNUM, 0, *aStream>>>(inBuff, outBuff, myMap);

  hipfftSetStream(*myPlan, *aStream);
  hipfftExecC2C(*myPlan, (hipfftComplex *)outBuff, (hipfftComplex *)outBuff, HIPFFT_BACKWARD);
}

