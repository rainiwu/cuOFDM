#include "hip/hip_runtime.h"
#include "Modulator.cuh"
#include <hipfft/hipfft.h>

__global__ void modulate(uint8_t* dBits, hipComplex* dMods, hipComplex* dMap) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  dMods[tid] = dMap[dBits[tid]];

}

void process(uint8_t* inBuff, hipComplex* outBuff, hipComplex* myMap, hipStream_t* aStream) {
  modulate<<<BATCH_SIZE/MOD_TNUM, MOD_TNUM, 0, *aStream>>>(inBuff, outBuff, myMap);

  hipfftHandle plan;
  hipfftPlan1d(&plan, BATCH_SIZE, HIPFFT_C2C, 1);
  hipfftSetStream(plan, *aStream);
  hipfftExecC2C(plan, (hipfftComplex *)outBuff, (hipfftComplex *)outBuff, HIPFFT_BACKWARD);
  hipfftDestroy(plan);
}

