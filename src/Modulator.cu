#include "hip/hip_runtime.h"
#include "Modulator.cuh"
#include <cstdio>

__global__ void modulate(uint8_t* dBits, hipComplex* dMods, hipComplex* dMap) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= BATCH_SIZE) printf("mistake in modulate"); 
  dMods[tid] = dMap[dBits[tid]];
}

void processBitBatch(uint8_t* hBitBatch, uint8_t* dBitBatch, hipComplex* hModBatch, hipComplex* dModBatch, hipComplex* dMap){
  hipMemcpy(dBitBatch, hBitBatch, sizeof(uint8_t)*BATCH_SIZE, hipMemcpyHostToDevice);
  modulate<<<BATCH_SIZE / TNUM, TNUM>>>(dBitBatch, dModBatch, dMap);
  hipDeviceSynchronize();
  hipMemcpy(hModBatch, dModBatch, sizeof(hipComplex)*BATCH_SIZE, hipMemcpyDeviceToHost);
}
