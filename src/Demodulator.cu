#include "hip/hip_runtime.h"
#include "Demodulator.cuh"
#include <hipfft/hipfft.h>

__global__ void demap(hipComplex* dMods, uint8_t* dBits, hipComplex* dMap) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // revert scaling
  dMods[tid].x = dMods[tid].x / BATCH_SIZE;
  dMods[tid].y = dMods[tid].y / BATCH_SIZE;

  uint8_t i = 0;
  do {
    if((dMods[tid].x == dMap[i].x && dMods[tid].y == dMap[i].y) ||
        (dMods[tid].x - dMap[i].x < 0.01 && dMods[tid].y - dMap[i].y < 0.01)) {
      dBits[tid] = i;
      break;
    }
  } while (++i != 0);

}

void demod(hipComplex* inBuff, uint8_t *outBuff, hipComplex* dMap, hipStream_t* aStream) {
  hipfftHandle plan;
  hipfftPlan1d(&plan, BATCH_SIZE, HIPFFT_C2C, 1);
  hipfftSetStream(plan, *aStream);
  hipfftExecC2C(plan, (hipfftComplex*)inBuff, (hipfftComplex*)inBuff, HIPFFT_FORWARD);
  hipfftDestroy(plan);

  demap<<<BATCH_SIZE/DEMOD_TNUM, DEMOD_TNUM, 0, *aStream>>>(inBuff, outBuff, dMap);
}
