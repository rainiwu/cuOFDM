#include "hip/hip_runtime.h"
#include "test.cuh"
#include <cstdio>

__global__ void bitsToIq(hipComplex *dMap, uint8_t *dIn, hipComplex *dOut) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  dOut[tid] = dMap[dIn[tid]];
  printf("hello");
}

void callBitsToIq(hipComplex *dMap, uint8_t *dIn, hipComplex *dOut, size_t size) {
  bitsToIq<<<1, size>>>(dMap, dIn, dOut);
}

